
#include <hip/hip_runtime.h>
/*! \file VL_2D_cuda.cu
 *  \brief Definitions of the cuda 2D VL algorithm functions. */

#ifdef CUDA
#ifdef VL

#include<stdio.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"VL_2D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"h_correction_2D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_2D.h"


__global__ void Update_Conserved_Variables_2D_half(Real *dev_conserved, Real *dev_conserved_half, 
                                                   Real *dev_F_x, Real *dev_F_y, int nx, int ny,
                                                   int n_ghost, Real dx, Real dy, Real dt, Real gamma, int n_fields);


Real VL_Algorithm_2D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int ny, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real dt, int n_fields)
{

  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array
  //host_conserved0 contains the values at time n,
  //host_conserved1 will contain the values at time n+1

  #ifdef TIME
  // capture the start time
  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  float elapsedTime;
  #endif

  // dimensions of subgrid blocks
  int nx_s, ny_s; 
  int nz_s = 1; //number of cells in the subgrid block along z direction
  int x_off_s, y_off_s; // x and y offsets for subgrid block

  // total number of blocks needed
  int block_tot;    //total number of subgrid blocks (unsplit == 1)
  int block1_tot;   //total number of subgrid blocks in x direction
  int block2_tot;   //total number of subgrid blocks in y direction
  int remainder1;   //modulus of number of cells after block subdivision in x direction
  int remainder2;   //modulus of number of cells after block subdivision in y direction 

  // counter for which block we're on
  int block = 0;

  // calculate the dimensions for each subgrid block
  sub_dimensions_2D(nx, ny, n_ghost, &nx_s, &ny_s, &block1_tot, &block2_tot, &remainder1, &remainder2, n_fields);
  //printf("%d %d %d %d %d %d\n", nx_s, ny_s, block1_tot, block2_tot, remainder1, remainder2);
  block_tot = block1_tot*block2_tot;

  // number of cells in one subgrid block
  int BLOCK_VOL = nx_s*ny_s*nz_s;

  // define the dimensions for the 2D grid
  int  ngrid = (BLOCK_VOL + 2*TPB - 1) / (2*TPB);

  //number of blocks per 2-d grid  
  dim3 dim2dGrid(ngrid, 2, 1);

  //number of threads per 1-d block   
  dim3 dim1dBlock(TPB, 1, 1);

  // Set up pointers for the location to copy from and to
  Real *tmp1;
  Real *tmp2;

  // allocate buffer to copy conserved variable blocks from and to 
  Real *buffer;
  if (block_tot > 1) {
    if ( NULL == ( buffer = (Real *) malloc(n_fields*BLOCK_VOL*sizeof(Real)) ) ) {
      printf("Failed to allocate CPU buffer.\n");
    }
    tmp1 = buffer;
    tmp2 = buffer;
  }
  else {
    tmp1 = host_conserved0;
    tmp2 = host_conserved1;
  }

  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(2*ngrid*sizeof(Real));
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  Real *host_dt_array;
  host_dt_array = (Real *) malloc(2*ngrid*sizeof(Real));
  #endif  

  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved, *dev_conserved_half;
  // input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *F_x, *F_y;
  // arrays to hold the eta values for the H correction
  Real *eta_x, *eta_y, *etah_x, *etah_y;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;
  #ifdef COOLING_GPU
  // array of timesteps for dt calculation (cooling restriction)
  Real *dev_dt_array;
  #endif  

  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_conserved_half, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lx, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rx, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ly, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ry, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_x,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_y,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_x,   BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_y,   BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_x,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_y,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, 2*ngrid*sizeof(Real)) );
  #ifdef COOLING_GPU
  CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
  #endif    


  // START LOOP OVER SUBGRID BLOCKS HERE
  while (block < block_tot) {

    // copy the conserved variable block to the buffer
    host_copy_block_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved0, buffer, n_fields);

    // calculate the global x and y offsets of this subgrid block
    // (only needed for gravitational potential)
    get_offsets_2D(nx_s, ny_s, n_ghost, x_off, y_off, block, block1_tot, block2_tot, remainder1, remainder2, &x_off_s, &y_off_s);    

    // copy the conserved variables onto the GPU
    CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );


    // Step 1: Use PCM reconstruction to put conserved variables into interface arrays
    PCM_Reconstruction_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, nx_s, ny_s, n_ghost, gama, n_fields);
    CudaCheckError();


    // Step 2: Calculate first-order upwind fluxes 
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    #endif
    #ifdef HLLC 
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    #endif
    CudaCheckError();


    // Step 3: Update the conserved variables half a timestep 
    Update_Conserved_Variables_2D_half<<<dim2dGrid,dim1dBlock>>>(dev_conserved, dev_conserved_half, F_x, F_y, nx_s, ny_s, n_ghost, dx, dy, 0.5*dt, gama, n_fields);
    CudaCheckError();


    // Step 4: Construct left and right interface values using updated conserved variables
    #ifdef PLMP
    PLMP_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PLMP_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    #endif
    #ifdef PLMC
    PLMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PLMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);    
    #endif
    #ifdef PPMP
    PPMP_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PPMP_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    #endif //PPMP
    #ifdef PPMC
    PPMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PPMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    #endif //PPMC
    CudaCheckError();


    #ifdef H_CORRECTION
    // Step 4.5: Calculate eta values for H correction
    calc_eta_x_2D<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, n_ghost, gama);
    calc_eta_y_2D<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();
    // and etah values for each interface
    calc_etah_x_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_x, nx_s, ny_s, n_ghost);
    calc_etah_y_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_y, nx_s, ny_s, n_ghost);
    CudaCheckError();
    #endif


    // Step 5: Calculate the fluxes again
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    #endif
    #ifdef HLLC 
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    #endif
    CudaCheckError();


    // Step 6: Update the conserved variable array
    Update_Conserved_Variables_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, nx_s, ny_s, x_off_s, y_off_s, n_ghost, dx, dy, xbound, ybound, dt, gama, n_fields);
    CudaCheckError();


    #ifdef DE
    Sync_Energies_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, gama, n_fields);
    CudaCheckError();
    #endif        


    // Apply cooling
    #ifdef COOLING_GPU
    cooling_kernel<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields, dt, gama, dev_dt_array);
    CudaCheckError();
    #endif


    // Step 7: Calculate the next timestep
    Calc_dt_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, dx, dy, dev_dti_array, gama);
    CudaCheckError();  


    // copy the conserved variable array back to the CPU
    CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );

    // copy the updated conserved variable array back into the host_conserved array on the CPU
    host_return_block_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved1, buffer, n_fields);


    // copy the dti array onto the CPU
    CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, 2*ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the maximum inverse dt for this subgrid block
    for (int i=0; i<2*ngrid; i++) {
      max_dti = fmax(max_dti, host_dti_array[i]);
    }
    #ifdef COOLING_GPU
    // copy the dt array from cooling onto the CPU
    CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the minimum dt for this subgrid block
    for (int i=0; i<2*ngrid; i++) {
      min_dt = fmin(min_dt, host_dt_array[i]);
    }  
    if (min_dt < C_cfl/max_dti) {
      max_dti = C_cfl/min_dt;
    }
    #endif

    // add one to the counter
    block++;

  }


  // free the CPU memory
  free(host_dti_array);
  if (block_tot > 1) free(buffer);
  #ifdef COOLING_GPU
  free(host_dt_array);  
  #endif    

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(dev_conserved_half);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(eta_x);
  cudaFree(eta_y);
  cudaFree(etah_x);
  cudaFree(etah_y);
  cudaFree(dev_dti_array);
  #ifdef COOLING_GPU
  cudaFree(dev_dt_array);
  #endif

  // return the maximum inverse timestep
  return max_dti;

}


__global__ void Update_Conserved_Variables_2D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y, int nx, int ny, int n_ghost, Real dx, Real dy, Real dt, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  int imo, jmo;

  Real dtodx = dt/dx;
  Real dtody = dt/dy;

  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;

  #ifdef DE
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, P;
  int ipo, jpo;
  #endif


  // all threads but one outer ring of ghost cells 
  if (xid > 0 && xid < nx-1 && yid > 0 && yid < ny-1)
  {
    imo = xid-1 + yid*nx;
    jmo = xid + (yid-1)*nx;
    #ifdef DE
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    //if (d < 0.0 || d != d) printf("Negative density before half step update.\n");
    //if (P < 0.0) printf("%d Negative pressure before half step update.\n", id);
    ipo = xid+1 + yid*nx;
    jpo = xid + (yid+1)*nx;
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo]; 
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo]; 
    #endif
    // update the conserved variable array
    dev_conserved_half[            id] = dev_conserved[            id] 
                                       + dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                       + dtody * (dev_F_y[            jmo] - dev_F_y[            id]);
    dev_conserved_half[  n_cells + id] = dev_conserved[  n_cells + id] 
                                       + dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]) 
                                       + dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_conserved_half[2*n_cells + id] = dev_conserved[2*n_cells + id] 
                                       + dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]) 
                                       + dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]); 
    dev_conserved_half[3*n_cells + id] = dev_conserved[3*n_cells + id] 
                                       + dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                       + dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_conserved_half[4*n_cells + id] = dev_conserved[4*n_cells + id] 
                                       + dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                       + dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved_half[(5+i)*n_cells + id] = dev_conserved[(5+i)*n_cells + id] 
                                         + dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                         + dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved_half[(n_fields-1)*n_cells + id] = dev_conserved[(n_fields-1)*n_cells + id] 
                                       + dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                       + dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                                       + 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo));
    #endif
                                       
  } 
}




#endif //VL
#endif //CUDA

