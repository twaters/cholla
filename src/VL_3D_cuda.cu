
#include <hip/hip_runtime.h>
/*! \file VL_3D_cuda.cu
 *  \brief Definitions of the cuda 3D VL algorithm functions. */

#ifdef CUDA
#ifdef VL

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"VL_3D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_vl_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_vl_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"h_correction_3D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_3D.h"


__global__ void Update_Conserved_Variables_3D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma);



Real VL_Algorithm_3D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int ny, int nz, int x_off, int y_off, int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real dt)
{

  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array

  int n_fields = 5;
  #ifdef DE
  n_fields++;
  #endif

  // number of cells
  int BLOCK_VOL = nx*ny*nz;

  // define the dimensions for the 1D grid
  int  ngrid = (BLOCK_VOL + TPB - 1) / TPB;

  //number of blocks per 1-d grid  
  dim3 dim1dGrid(ngrid, 1, 1);

  //number of threads per 1-d block   
  dim3 dim1dBlock(TPB, 1, 1);


  // Set up pointers for the location to copy from and to
  Real *tmp1 = host_conserved0;
  Real *tmp2 = host_conserved1;


  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(ngrid*sizeof(Real));
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  Real *host_dt_array;
  host_dt_array = (Real *) malloc(ngrid*sizeof(Real));
  #endif  

  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved, *dev_conserved_half;
  // input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *Q_Lz, *Q_Rz, *F_x, *F_y, *F_z;
  // arrays to hold the eta values for the H correction
  Real *eta_x, *eta_y, *eta_z, *etah_x, *etah_y, *etah_z;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;
  #ifdef COOLING_GPU
  // array of timesteps for dt calculation (cooling restriction)
  Real *dev_dt_array;
  #endif  

  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_conserved_half, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lx,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rx,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ly,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ry,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lz,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rz,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_x,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_y,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_z,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_x,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_y,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_z,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_x, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_y, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_z, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
  #ifdef COOLING_GPU
  CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
  #endif  

  // zero the GPU arrays
  cudaMemset(dev_conserved, 0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(dev_conserved_half, 0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Lx,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Rx,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Ly,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Ry,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Lz,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(Q_Rz,  0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(F_x,   0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(F_y,   0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(F_z,   0, n_fields*BLOCK_VOL*sizeof(Real));
  cudaMemset(eta_x,  0, BLOCK_VOL*sizeof(Real));
  cudaMemset(eta_y,  0, BLOCK_VOL*sizeof(Real));
  cudaMemset(eta_z,  0, BLOCK_VOL*sizeof(Real));
  cudaMemset(etah_x, 0, BLOCK_VOL*sizeof(Real));
  cudaMemset(etah_y, 0, BLOCK_VOL*sizeof(Real));
  cudaMemset(etah_z, 0, BLOCK_VOL*sizeof(Real));
  cudaMemset(dev_dti_array, 0, ngrid*sizeof(Real));  
  CudaCheckError();


  // copy the conserved variables onto the GPU
  CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );
  

  // Step 1: Use PCM reconstruction to put primitive variables into interface arrays
  PCM_Reconstruction_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, gama);
  CudaCheckError();


  // Step 2: Calculate first-order upwind fluxes 
  #ifdef EXACT
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0);
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1);
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2);
  #endif //EXACT
  #ifdef ROE
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, etah_x, 0);
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, etah_y, 1);
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, etah_z, 2);
  #endif //ROE
  #ifdef HLLC 
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, etah_x, 0);
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, etah_y, 1);
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, etah_z, 2);
  #endif //HLLC
  CudaCheckError();


  // Step 3: Update the conserved variables half a timestep 
  Update_Conserved_Variables_3D_half<<<dim1dGrid,dim1dBlock>>>(dev_conserved, dev_conserved_half, F_x, F_y, F_z, nx, ny, nz, n_ghost, dx, dy, dz, 0.5*dt, gama);
  CudaCheckError();
  // Apply cooling
  #ifdef COOLING_GPU
  //cooling_kernel<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, nx, ny, nz, n_ghost, 0.5*dt, gama);
  //CudaCheckError();
  #endif



  // Step 4: Construct left and right interface values using updated conserved variables
  #ifdef PCM
  PCM_Reconstruction_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, gama);
  #endif
  #ifdef PLMP
  PLMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, gama, 0);
  PLMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, gama, 1);
  PLMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, gama, 2);
  #endif //PLMP 
  #ifdef PLMC
  PLMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  PLMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt, gama, 1);
  PLMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt, gama, 2);  
  #endif
  #ifdef PPMP
  PPMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, gama, 0);
  PPMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, gama, 1);
  PPMP_VL<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, gama, 2);
  #endif //PPMP
  #ifdef PPMC
  PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt, gama, 1);
  PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt, gama, 2);
  #endif //PPMC
  CudaCheckError();
  

  #ifdef H_CORRECTION
  // Step 4.5: Calculate eta values for H correction
  calc_eta_x_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx, ny, nz, n_ghost, gama);
  calc_eta_y_3D<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx, ny, nz, n_ghost, gama);
  calc_eta_z_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, eta_z, nx, ny, nz, n_ghost, gama);
  CudaCheckError();
  // and etah values for each interface
  calc_etah_x_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_x, nx, ny, nz, n_ghost);
  calc_etah_y_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_y, nx, ny, nz, n_ghost);
  calc_etah_z_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_z, nx, ny, nz, n_ghost);
  CudaCheckError();
  #endif //H_CORRECTION


  // Step 5: Calculate the fluxes again
  #ifdef EXACT
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0);
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1);
  Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2);
  #endif //EXACT
  #ifdef ROE
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, etah_x, 0);
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, etah_y, 1);
  Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, etah_z, 2);
  #endif //ROE
  #ifdef HLLC 
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, etah_x, 0);
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, etah_y, 1);
  Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, etah_z, 2);
  #endif //HLLC
  CudaCheckError();


  // Step 6: Update the conserved variable array
  Update_Conserved_Variables_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, F_z, nx, ny, nz, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound, zbound, dt, gama);
  CudaCheckError();

  #ifdef DE
  Sync_Energies_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx, ny, nz, n_ghost, gama);
  CudaCheckError();
  #endif

  // Apply cooling
  #ifdef COOLING_GPU
  //cooling_kernel<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx, ny, nz, n_ghost, dt, gama);
  cooling_kernel<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx, ny, nz, n_ghost, dt, gama, dev_dt_array);  
  CudaCheckError();
  #endif

  
  // Step 7: Calculate the next time step
  Calc_dt_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx, ny, nz, n_ghost, dx, dy, dz, dev_dti_array, gama);
  CudaCheckError();


  // copy the updated conserved variable array back to the CPU
  CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );


  // copy the dti array onto the CPU
  CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // iterate through to find the maximum inverse dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    max_dti = fmax(max_dti, host_dti_array[i]);
  }
  #ifdef COOLING_GPU
  // copy the dt array from cooling onto the CPU
  CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // iterate through to find the minimum dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    min_dt = fmin(min_dt, host_dt_array[i]);
  }  
  //printf("%f %f\n", min_dt, 0.3/max_dti); 
  if (min_dt < 0.3/max_dti) {
    printf("%f %f\n", min_dt, 0.3/max_dti); 
    min_dt = fmax(min_dt, 1.0);
    max_dti = 0.3/min_dt;
  }
  #endif
  //max_dti = fmin(max_dti, 0.3);


  // free CPU memory
  free(host_dti_array);  
  #ifdef COOLING_GPU
  free(host_dt_array);  
  #endif  

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(dev_conserved_half);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(Q_Lz);
  cudaFree(Q_Rz);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(F_z);
  cudaFree(eta_x);
  cudaFree(eta_y);
  cudaFree(eta_z);
  cudaFree(etah_x);
  cudaFree(etah_y);
  cudaFree(etah_z);
  cudaFree(dev_dti_array);
  #ifdef COOLING_GPU
  cudaFree(dev_dt_array);
  #endif


  // return the maximum inverse timestep
  return max_dti;

}


__global__ void Update_Conserved_Variables_3D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma)
{
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  int n_cells = nx*ny*nz;

  // get a global thread ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;
  int id = xid + yid*nx + zid*nx*ny;

  int imo = xid-1 + yid*nx + zid*nx*ny;
  int jmo = xid + (yid-1)*nx + zid*nx*ny;
  int kmo = xid + yid*nx + (zid-1)*nx*ny;

  #ifdef DE
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, vz_kmo, vz_kpo, P;
  int ipo, jpo, kpo;
  #endif

  // threads corresponding to all cells except outer ring of ghost cells do the calculation
  if (xid > 0 && xid < nx-1 && yid > 0 && yid < ny-1 && zid > 0 && zid < nz-1)
  {
    #ifdef DE
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    //if (d < 0.0 || d != d) printf("Negative density before half step update.\n");
    //if (P < 0.0) printf("%d Negative pressure before half step update.\n", id);
    ipo = xid+1 + yid*nx + zid*nx*ny;
    jpo = xid + (yid+1)*nx + zid*nx*ny;
    kpo = xid + yid*nx + (zid+1)*nx*ny;
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo]; 
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo]; 
    vz_kmo = dev_conserved[3*n_cells + kmo] / dev_conserved[kmo]; 
    vz_kpo = dev_conserved[3*n_cells + kpo] / dev_conserved[kpo]; 
    #endif
  
    // update the conserved variable array
    dev_conserved_half[            id] = dev_conserved[            id]
                                       + dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                       + dtody * (dev_F_y[            jmo] - dev_F_y[            id])
                                       + dtodz * (dev_F_z[            kmo] - dev_F_z[            id]);
    dev_conserved_half[  n_cells + id] = dev_conserved[  n_cells + id] 
                                       + dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                                       + dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                                       + dtodz * (dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_conserved_half[2*n_cells + id] = dev_conserved[2*n_cells + id] 
                                       + dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                                       + dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                                       + dtodz * (dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_conserved_half[3*n_cells + id] = dev_conserved[3*n_cells + id] 
                                       + dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                       + dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                                       + dtodz * (dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_conserved_half[4*n_cells + id] = dev_conserved[4*n_cells + id] 
                                       + dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                       + dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                                       + dtodz * (dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef DE
    dev_conserved_half[5*n_cells + id] = dev_conserved[5*n_cells + id] 
                                       + dtodx * (dev_F_x[5*n_cells + imo] - dev_F_x[5*n_cells + id])
                                       + dtody * (dev_F_y[5*n_cells + jmo] - dev_F_y[5*n_cells + id])
                                       + dtodz * (dev_F_z[5*n_cells + kmo] - dev_F_z[5*n_cells + id])
                                       + 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));
    #endif
    //if (dev_conserved_half[id] < 0.0 || dev_conserved_half[id] != dev_conserved_half[id] || dev_conserved_half[4*n_cells+id] < 0.0 || dev_conserved_half[4*n_cells+id] != dev_conserved_half[4*n_cells+id]) {
      //printf("%3d %3d %3d Thread crashed in half step update. d: %e E: %e\n", xid, yid, zid, dev_conserved_half[id], dev_conserved_half[4*n_cells+id]);
    //}    

  }

}




#endif //VL
#endif //CUDA
